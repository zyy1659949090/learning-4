
/* 
   xor_pcases.cu
   Implementation of a XOR neural network in CUDA, 
   calculating output of many input cases in parallel.

   Andrei de A. Formiga, 2012-03-31
*/


#include <hip/hip_runtime.h>
#include <stdio.h>


// weights for the hidden layer
float weights_h[] = { 0.5f, -1.0f, -1.0f,
                      -1.5f, 1.0f, 1.0f };

// weights for the output layer
float weights_o[] = { 0.5f, -1.0f, -1.0f };

// weight arrays for the device
float *dev_hw;
float *dev_ow;

// device input
float *dev_in;

// device hidden outputs
float *dev_hidden;

// device output
float *dev_out;

// inputs
float inputs[] = { 0.0f, 0.0f, 0.0f, 1.0f,
                   1.0f, 0.0f, 1.0f, 1.0f };

int ncases = 4;
int input_size = 2;

int hidden_size = 2;

// desired outputs
float outputs[] = { 0.0f, 1.0f, 1.0f, 0.0f };

// kernel for hidden layer
__global__ void calculate_hidden(float *dev_hw, float *input, float *hidden)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int input_ix = blockIdx.x * 2;  // 2 neurons in the previous layer
    int toff = threadIdx.x;
    float h;

    h = dev_hw[toff * 3] * 1.0f +
        dev_hw[toff * 3 + 1] * input[input_ix] +
        dev_hw[toff * 3 + 2] * input[input_ix+1];

    // threshold
    if (h > 0.0f)
        hidden[tid] = 1.0f;
    else
        hidden[tid] = 0.0;
}

// kernel for output layer
__global__ void calculate_output(float *dev_ow, float *hidden, float *output)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int hidden_ix = blockIdx.x * 2;  // 2 neurons in the previous layer
    int toff = threadIdx.x;    
    float o;

    o = dev_ow[toff] * 1.0f +
        dev_ow[toff+1] * hidden[hidden_ix] +
        dev_ow[toff+2] * hidden[hidden_ix+1];

    // threshold
    if (o > 0.0f)
        output[tid] = 1.0f;
    else
        output[tid] = 0.0f;
}

int main(int argc, char **argv)
{
    float out[ncases];

    printf("### XOR test (forward propagation)\n");
    
    hipMalloc((void**) &dev_hw, 6 * sizeof(float));
    hipMalloc((void**) &dev_ow, 3 * sizeof(float));
    hipMalloc((void**) &dev_in, ncases * input_size * sizeof(float));
    hipMalloc((void**) &dev_hidden, ncases * hidden_size * sizeof(float));
    hipMalloc((void**) &dev_out, ncases * sizeof(float));  // output size = 1
    
    hipMemcpy(dev_hw, weights_h, 6 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_ow, weights_o, 3 * sizeof(float), hipMemcpyHostToDevice);

    // try inputs
    hipMemcpy(dev_in, inputs, ncases * input_size * sizeof(float), hipMemcpyHostToDevice);
    calculate_hidden<<<4, 2>>>(dev_hw, dev_in, dev_hidden);
    calculate_output<<<4, 1>>>(dev_ow, dev_hidden, dev_out);
    hipMemcpy(out, dev_out, ncases * sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < ncases; ++i)
        printf("Input: %2.1f %2.1f -- Output: %f\n", inputs[input_size*i],
               inputs[input_size*i+1], out[i]);
    
    hipFree(dev_hw);
    hipFree(dev_ow);
    hipFree(dev_in);
    hipFree(dev_hidden);
    hipFree(dev_out);
    
    return 0;
}
